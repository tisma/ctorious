#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hiprand.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

void GPU_fill_rand(float* A, int nr_rows_A, int nr_cols_A) {
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

void gpu_blas_matix_mul(const float* A, const float* B, float* C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	const float alf = 1.0f;
	const float bet = 0.0f;
	const float* alpha = &alf;
	const float* beta = &bet;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	hipblasDestroy(handle);
}

void print_matrix(thrust::device_vector<float>& A) {
	for (int i = 0; i < A.size(); i++) {
		cout << A[i] << ' ';
	}
	cout << endl;
}

int main() {
	int nr_rows_A = 3, nr_cols_A = 3, nr_rows_B = 3, nr_cols_B = 3, nr_rows_C = 3, nr_cols_C = 3;

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A);
	thrust::device_vector<float> d_B(nr_rows_B * nr_cols_B);
	thrust::device_vector<float> d_C(nr_rows_C * nr_cols_C);

	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);

	cout << "A = " << endl;
	print_matrix(d_A);
	cout << "B = " << endl;
	print_matrix(d_B);

	gpu_blas_matix_mul(thrust::raw_pointer_cast(&d_A[0]), 
					   thrust::raw_pointer_cast(&d_B[0]), 
					   thrust::raw_pointer_cast(&d_C[0]), 
					   nr_rows_A, nr_cols_A, nr_cols_B);

	cout << "C = " << endl;
	print_matrix(d_C);

	return 0;
}
